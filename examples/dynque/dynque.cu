#include "hip/hip_runtime.h"
// TODO:
// 1) we can do a single persistence atomic counter
// 2) status array should be just char-byte
// 3) make each warp to wait for timeout and then - exit. Stop counting, when at least one warp has already exited

#include <cstdlib>
#include <iostream>
#include <time.h>
#include <tools/utils.h>
#include <unistd.h>

#include "queueDistLocks.cuh"
#include "queueShared.cuh"
#include "queuingPerProc.cuh"
#include "techniqueMegakernel.cuh"
#include "techniqueKernels.cuh"
#include "techniqueDynamicParallelism.cuh"
#include "segmentedStorage.cuh"

#include "procedureInterface.cuh"
#include "procinfoTemplate.cuh"

// TODO: specify regcount

namespace PersistenceTest
{
	static const uint timeout = 2;
	__device__ uint dCount;
	static uint* hFinish;
	__constant__ uint dFinish;

	class Task : public ::Procedure
	{
	public:
		static const int NumThreads = 32;
		static const bool ItemInput = false; // false results in a lvl 1 task
		static const int sharedMemory = 0; // shared memory requirements 

		typedef void* ExpectedData;
	
		template<class Q, class Context>
		static __device__ __inline__ void execute(int threadId, int numThreads, Q* queue, ExpectedData* data, volatile uint* shared)
		{
			// This kernel counts the number of warps that can be launched
			// by device as a single wavefront.
	
			// Count only while not finished
			if ((threadId == 0) && !dFinish)
				atomicAdd(&dCount, 1);

			// Wait till finish signal from host.	
			while (atomicCAS(&dFinish, 1, 1) != 1) { clock64(); }
		}

		template<class Q>
		__device__ __inline__ static void init(Q* q, int id)
		{
			q->template enqueueInitial<PersistenceTest::Task>(NULL);
		}
	};

	// Lets use a dist locks queue for each procedure, which can hold 96k elements
	template<class ProcInfo>
	class MyQueue : public PerProcedureQueueTyping<QueueDistLocksOpt_t, 96 * 1024, false>::Type<ProcInfo>
	{
	};

	typedef Megakernel::SimplePointed16336<MyQueue, ProcInfo<PersistenceTest::Task> > MyTechnique;

	// Find the maximum number of persistent tasks the device can carry.	
	static uint run()
	{
		hipDeviceProp_t deviceProp;
		CUDA_CHECKED_CALL(hipGetDeviceProperties(&deviceProp, 0));

		uint hCount = 0;
		uint zero = 0;
		hipStream_t stream1, stream2;
		CUDA_CHECKED_CALL(hipStreamCreate(&stream1));
		CUDA_CHECKED_CALL(hipStreamCreate(&stream2));
		for (int ntasks = 1, ntasks_max = deviceProp.maxGridSize[0]; ntasks < ntasks_max; ntasks *= 2)
		{
			// Reset counter.
			CUDA_CHECKED_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dCount), &zero, sizeof(uint)));
			
			// Reset finish marker.
			CUDA_CHECKED_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dFinish), &zero, sizeof(uint)));

			// Launch uberkernel.
			MyTechnique technique;
			technique.init();
			technique.insertIntoQueue<PersistenceTest::Task>(ntasks);

			technique.execute(0, stream1);
		
			// Make uberkernel to work for a while.
			usleep(1000000 * timeout);

			// Signal shut down to uberkernel.
			uint one = 1;
			/*CUDA_CHECKED_CALL(hipMemcpyAsync(hFinish, &one, sizeof(uint), hipMemcpyHostToDevice, stream2));
			CUDA_CHECKED_CALL(hipStreamSynchronize(stream2));*/
			CUDA_CHECKED_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(dFinish), &one, sizeof(uint), 0, hipMemcpyHostToDevice, stream2));
			CUDA_CHECKED_CALL(hipStreamSynchronize(stream2));
		
			CUDA_CHECKED_CALL(hipStreamSynchronize(stream1));
		
			uint hCountCurrent;
			CUDA_CHECKED_CALL(hipMemcpyFromSymbol(&hCountCurrent, HIP_SYMBOL(dCount), sizeof(uint)));
			printf("# persistent warps: %u\n", hCountCurrent);
			if (hCountCurrent <= hCount)
				break;
			hCount = hCountCurrent;
		}
		CUDA_CHECKED_CALL(hipHostFree(hFinish));
		printf("# max persistent warps: %u\n", hCount);
		CUDA_CHECKED_CALL(hipStreamDestroy(stream1));
		CUDA_CHECKED_CALL(hipStreamDestroy(stream2));
		
		return hCount;
	}
}

struct TaskInfo
{
	uint id;
	uint* ready;
};

static uint* status;

class Task : public ::Procedure
{
public:
	static const int NumThreads = 32;
	static const bool ItemInput = false; // false results in a lvl 1 task
	static const int sharedMemory = 0; // shared memory requirements 
	
	typedef TaskInfo ExpectedData;

	template<class Q, class Context>
	static __device__ __inline__ void execute(int threadId, int numThreads, Q* queue, ExpectedData* ptask, volatile uint* shared)
	{
		while (!atomicCAS(ptask->ready, 1, 0)) { clock64(); }
		
		if (threadId == 0)
			printf("Executing task %04u\n", ptask->id);
	}

	template<class Q>
	__device__ __inline__ static void init(Q* q, int id)
	{
		TaskInfo task;
		task.id = id;
		task.ready = &status[id];
		q->template enqueueInitial<Task>(task);
	}
};

int main(int argc, char** argv)
{
	using namespace std;

	{
		int count;
		CUDA_CHECKED_CALL(hipGetDeviceCount(&count));
		if (!count)
		{
			cerr << "No CUDA devices available" << endl;
			return -1;
		}
		hipDeviceProp_t deviceProp;
		CUDA_CHECKED_CALL(hipGetDeviceProperties(&deviceProp, 0));
		cout << "Using device: " << deviceProp.name << endl;
	}

	// Find the maximum number of persistent tasks the device can carry.	
	uint count = PersistenceTest::run();

#if 0
	//
	// 2) Launch the maximum number of persistent tasks the device can carry.
	//
	{
		// Create array of task readiness locks. These will be atomically
		// CAS-ed by host and by whippletree.
		uint* hstatus = NULL;
		CUDA_CHECKED_CALL(hipHostAlloc(&hstatus, sizeof(uint) * ntasks, hipHostMallocMapped));
		CUDA_CHECKED_CALL(hipMemset(hstatus, 0, sizeof(uint) * ntasks));
		CUDA_CHECKED_CALL(hipMemcpyToSymbol(HIP_SYMBOL(status), &hstatus, sizeof(uint*)));

		MyTechnique technique;
		technique.init();
		technique.insertIntoQueue<Task>(ntasks);

		hipStream_t stream;
		CUDA_CHECKED_CALL(hipStreamCreate(&stream));
		hipEvent_t a, b;
		CUDA_CHECKED_CALL(hipEventCreate(&a));
		CUDA_CHECKED_CALL(hipEventCreate(&b));
		CUDA_CHECKED_CALL(hipEventRecord(a, stream));

		technique.execute(0, stream);

		CUDA_CHECKED_CALL(hipEventRecord(b, stream));
		CUDA_CHECKED_CALL(hipEventSynchronize(b));
		float time;
		CUDA_CHECKED_CALL(hipEventElapsedTime(&time, a, b));
		time /= 1000.0;
		CUDA_CHECKED_CALL(hipEventDestroy(a));
		CUDA_CHECKED_CALL(hipEventDestroy(b));
		CUDA_CHECKED_CALL(hipStreamDestroy(stream));

		CUDA_CHECKED_CALL(hipHostFree(hstatus));
	}
#endif

	return 0;
}

