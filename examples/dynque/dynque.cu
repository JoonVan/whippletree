// TODO:
// 1) Extend megakernel interface to run infinitely, waiting for tasks
// 2) Implement tasks queuing from host to device (requires 2-way dynamic data exchange?).
// 3) Develop system to determine pointers of task device functions

#include <cstdlib>
#include <iostream>
#include <time.h>
#include <tools/utils.h>
#include <unistd.h>

#include "queueDistLocks.cuh"
#include "queueShared.cuh"
#include "queuingPerProc.cuh"
#include "techniqueMegakernel.cuh"
#include "techniqueKernels.cuh"
#include "techniqueDynamicParallelism.cuh"
#include "segmentedStorage.cuh"

#include "procedureInterface.cuh"
#include "procinfoTemplate.cuh"

typedef void (*DynamicTaskFunction)(int threadId, int numThreads, void* data, volatile uint* shared);

struct DynamicTaskInfo
{
	DynamicTaskFunction func;
	void* data;
};

namespace
{
	__constant__ bool finish;
}

class DynamicTaskManager
{
	hipStream_t stream1, stream2;
	bool* address;
	
public :

	class Task : public ::Procedure
	{
	public:
		static const int NumThreads = 32;
		static const bool ItemInput = false; // false results in a lvl 1 task
		static const int sharedMemory = 0; // shared memory requirements 

		typedef DynamicTaskInfo ExpectedData;
	
		template<class Q, class Context>
		static __device__ __inline__ void execute(int threadId, int numThreads, Q* queue, ExpectedData* data, volatile uint* shared)
		{
			// Execute given task with the given argument.
			DynamicTaskInfo* task = (DynamicTaskInfo*)data;
			task->func(threadId, numThreads, task->data, shared);
		}

		template<class Q>
		__device__ __inline__ static void init(Q* q, int id)
		{
			// Not supposed to have any initial queue.
			__trap();
		}
	};

	// Lets use a dist locks queue for each procedure, which can hold 96k elements
	typedef PerProcedureQueueTyping<QueueDistLocksOpt_t, 96 * 1024, false> TQueue;

	template<class ProcInfo>
	class MyQueue : public TQueue::Type<ProcInfo>
	{
	public :
		static const int globalMaintainMinThreads = 1;
		
		__inline__ __device__ void globalMaintain()
		{
			if (threadIdx.x == 0)
			{
				printf("Insert something into queue!\n");

				//TQueue::Type<ProcInfo>::template enqueue<Task>(NULL);
			}			 
		}
	};

private :

	typedef Megakernel::SimplePointed16336<MyQueue, ProcInfo<Task>, void, Megakernel::ShutdownIndicator> MyTechnique;

	MyTechnique technique;

public :

	void start()
	{
		// Initialize finishing marker with "false" to make uberkernel
		// to run infinitely.
		bool value = false;
		CUDA_CHECKED_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(finish), &value, sizeof(bool), 0, hipMemcpyHostToDevice, stream2));
		CUDA_CHECKED_CALL(hipStreamSynchronize(stream2));

		// Start megakernel in a dedicated stream.
		technique.init();
		technique.execute(0, stream1, address);
	}
	
	void stop()
	{
		// Signal shut down to uberkernel.
		bool value = true;
		CUDA_CHECKED_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(finish), &value, sizeof(bool), 0, hipMemcpyHostToDevice, stream2));
		CUDA_CHECKED_CALL(hipStreamSynchronize(stream2));
		
		// Wait for uberkernel to finish.
		CUDA_CHECKED_CALL(hipStreamSynchronize(stream1));
	}

	void EnqueueTask()
	{
	}

	void EnqueueTaskAsync()
	{
	}

	DynamicTaskManager()
	{
		// Create two streams: one for megakernel, and another one -
		// for finish indicator.
		CUDA_CHECKED_CALL(hipStreamCreate(&stream1));
		CUDA_CHECKED_CALL(hipStreamCreate(&stream2));
		
		// Determine address of finishing marker to supply it into
		// the technique.
		CUDA_CHECKED_CALL(hipGetSymbolAddress((void**)&address, finish));
	}

	~DynamicTaskManager()
	{
		// Destroy streams.
		CUDA_CHECKED_CALL(hipStreamDestroy(stream1));
		CUDA_CHECKED_CALL(hipStreamDestroy(stream2));
	}
};

int main(int argc, char** argv)
{
	using namespace std;

	{
		int count;
		CUDA_CHECKED_CALL(hipGetDeviceCount(&count));
		if (!count)
		{
			cerr << "No CUDA devices available" << endl;
			return -1;
		}
		hipDeviceProp_t deviceProp;
		CUDA_CHECKED_CALL(hipGetDeviceProperties(&deviceProp, 0));
		cout << "Using device: " << deviceProp.name << endl;
	}

	DynamicTaskManager dtm;
	
	dtm.start();

	// Make uberkernel to work for a while.
	uint timeout = 10;
	usleep(1000000 * timeout);

	dtm.stop();

	return 0;
}

