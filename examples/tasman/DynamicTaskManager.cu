#include "DynamicTaskManager.h"

namespace
{
	__constant__ bool finish;
}

DynamicTask::~DynamicTask()
{
	CUDA_CHECKED_CALL(hipFree(info));
}

DynamicTaskManager::DynamicTaskManager()
{
	// Create two streams: one for megakernel, and another one -
	// for finish indicator.
	CUDA_CHECKED_CALL(hipStreamCreate(&stream1));
	CUDA_CHECKED_CALL(hipStreamCreate(&stream2));
	
	// Determine address of finishing marker to supply it into
	// the technique.
	CUDA_CHECKED_CALL(hipGetSymbolAddress((void**)&address, finish));
}

DynamicTaskManager::~DynamicTaskManager()
{
	// Destroy streams.
	CUDA_CHECKED_CALL(hipStreamDestroy(stream1));
	CUDA_CHECKED_CALL(hipStreamDestroy(stream2));
}

DynamicTaskManager& DynamicTaskManager::get()
{
	static DynamicTaskManager dtm;
	return dtm;
}

void DynamicTaskManager::start()
{
	// Initialize finishing marker with "false" to make uberkernel
	// to run infinitely.
	bool value = false;
	CUDA_CHECKED_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(finish), &value, sizeof(bool), 0, hipMemcpyHostToDevice, stream2));
	CUDA_CHECKED_CALL(hipStreamSynchronize(stream2));

	// Start megakernel in a dedicated stream.
	technique.init();
	technique.execute(0, stream1, address);
}

void DynamicTaskManager::stop()
{
	// Signal shut down to uberkernel.
	bool value = true;
	CUDA_CHECKED_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(finish), &value, sizeof(bool), 0, hipMemcpyHostToDevice, stream2));
	CUDA_CHECKED_CALL(hipStreamSynchronize(stream2));
	
	// Wait for uberkernel to finish.
	CUDA_CHECKED_CALL(hipStreamSynchronize(stream1));
}

void DynamicTaskManager::enqueue(const DynamicTask* task, void* data) const
{
	// Copy data to device memory.
	CUDA_CHECKED_CALL(hipMemcpyAsync(&task->info->data, &data, sizeof(void*), hipMemcpyHostToDevice, stream2));
	
	// Wait until queue gets empty.
	while (true)
	{
		DynamicTaskInfo* busy = NULL;
		CUDA_CHECKED_CALL(hipMemcpyFromSymbolAsync(&busy, HIP_SYMBOL(submission), sizeof(DynamicTaskInfo*), 0, hipMemcpyDeviceToHost, stream2));
		CUDA_CHECKED_CALL(hipStreamSynchronize(stream2));
		if (!busy) break;
	}

	// Submit task into queue.
	CUDA_CHECKED_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(submission), &task->info, sizeof(DynamicTaskInfo*), 0, hipMemcpyHostToDevice, stream2));
	CUDA_CHECKED_CALL(hipStreamSynchronize(stream2));
}

