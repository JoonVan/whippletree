#include "DynamicTaskManager.h"

extern "C" void dynamicTaskManagerStart(hipStream_t stream);

namespace tasman
{
	DynamicTask::~DynamicTask()
	{
		CHECKED_CALL(hipFree(info));
	}

	DynamicTaskManager::DynamicTaskManager() : started(false)
	{
		// Create two streams: one for megakernel, and another one -
		// for finish indicator.
		CHECKED_CALL(hipStreamCreate(&stream1));
		CHECKED_CALL(hipStreamCreate(&stream2));
	
		// Determine address of finishing marker to supply it into
		// the technique.
		CHECKED_CALL(hipGetSymbolAddress((void**)&address, finish));
	}

	DynamicTaskManager::~DynamicTaskManager()
	{
		// Destroy streams.
		CHECKED_CALL(hipStreamDestroy(stream1));
		CHECKED_CALL(hipStreamDestroy(stream2));
	}

	DynamicTaskManager& DynamicTaskManager::get()
	{
		static DynamicTaskManager dtm;
		return dtm;
	}

	void DynamicTaskManager::start()
	{
		if (started) return;
		started = true;

		// Initialize finishing marker with "false" to make uberkernel
		// to run infinitely.
		int value = 0;
		CHECKED_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(finish), &value, sizeof(int), 0, hipMemcpyHostToDevice, stream2));
		CHECKED_CALL(hipStreamSynchronize(stream2));
	
		dynamicTaskManagerStart(stream1);
	}

	void DynamicTaskManager::stop()
	{
		// Wait until queue gets empty.
		while (true)
		{
			DynamicTaskInfo* busy = NULL;
			CHECKED_CALL(hipMemcpyFromSymbolAsync(&busy, HIP_SYMBOL(submission), sizeof(DynamicTaskInfo*), 0, hipMemcpyDeviceToHost, stream2));
			CHECKED_CALL(hipStreamSynchronize(stream2));
			if (!busy) break;
		}

		// Signal shut down to uberkernel.
		int value = 1;
		CHECKED_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(finish), &value, sizeof(int), 0, hipMemcpyHostToDevice, stream2));
		CHECKED_CALL(hipStreamSynchronize(stream2));
	
		// Wait for uberkernel to finish.
		CHECKED_CALL(hipStreamSynchronize(stream1));
	
		started = false;
	}

	void DynamicTaskManager::enqueue(const DynamicTask* task, void* data) const
	{	
		// Wait until queue gets empty.
		while (true)
		{
			DynamicTaskInfo* busy = NULL;
			CHECKED_CALL(hipMemcpyFromSymbolAsync(&busy, HIP_SYMBOL(submission), sizeof(DynamicTaskInfo*), 0, hipMemcpyDeviceToHost, stream2));
			CHECKED_CALL(hipStreamSynchronize(stream2));
			if (!busy) break;
		}

		// Copy data to device memory.
		CHECKED_CALL(hipMemcpyAsync(&task->info->data, &data, sizeof(void*), hipMemcpyHostToDevice, stream2));

		// Submit task into queue.
		CHECKED_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(submission), &task->info, sizeof(DynamicTaskInfo*), 0, hipMemcpyHostToDevice, stream2));
		CHECKED_CALL(hipStreamSynchronize(stream2));
	}
}

